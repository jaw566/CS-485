
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <stdint.h>

using namespace std;

void warmUpGPU();

int main( int argc, char **argv )
{
	hipError_t error_code;
	unsigned long long num_items = atoll( argv[ 1 ] );
	unsigned long long upper_bound = atoll( argv[ 2 ] );

	warmUpGPU();
	const int num_trials = 3;
	const int STEP_SIZE = 125000000;
	int outer_index = 0;

		while( num_items < upper_bound )
		{

			for( outer_index = 0; outer_index < num_trials; outer_index++ )
			{
				char *dev_A = NULL;
				char *host_A = (char*) malloc( sizeof( char ) * num_items );

				int index = 0;
				for( index = 0; index < num_items - 1; index++ )
				{
					host_A[ index ] = 'A';
				}

				host_A[ num_items - 1 ] = '\0';

				error_code = hipMalloc( (char **) &dev_A, sizeof( char ) * num_items );
				hipDeviceSynchronize();

				if( error_code != hipSuccess )
				{
					cout << "Error allocating on device" << endl;
				}

				error_code = hipMemcpy( dev_A, host_A, sizeof( char ) * num_items, hipMemcpyHostToDevice );

				hipDeviceSynchronize();


				free( host_A );
				hipFree( dev_A );
				hipDeviceSynchronize();
			}

			num_items += STEP_SIZE;
			printf( "Doing a thing\n" );

		}


	return EXIT_SUCCESS;
}

__global__ void warmup( unsigned int *tmp )
{
    if( threadIdx.x == 0 )
        {
            *tmp = 555;
        }
    return;
}

void warmUpGPU()
{
    printf( "Warming up GPU for time trialing...\n" );

    unsigned int *dev_tmp;
    unsigned int *tmp;

    hipError_t errCode = hipSuccess;


    tmp = (unsigned int *) malloc( sizeof( unsigned int ) );
    errCode = hipMalloc( (unsigned int **) &dev_tmp, sizeof( unsigned int ) );

    if( errCode != hipSuccess )
        {
            cout << "Error: dev_tmp error with code " << errCode << endl;
        }

    warmup<<<1,256>>>(dev_tmp);

    //copy data from device to host 
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
    {
        cout << "Error: getting tmp result form GPU error with code " << errCode << endl; 
	}

	hipDeviceSynchronize();

	printf("tmp (changed to 555 on GPU): %d\n",*tmp);

    hipFree(dev_tmp);

    return;

}
